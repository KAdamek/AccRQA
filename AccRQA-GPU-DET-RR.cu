#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "debug.h"
#include "timer.h"
#include "utils_cuda.h"

#include <stdio.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <iomanip> 
#include <vector>

#include "GPU_reduction.cuh"
#include "AccRQA_metrics.cuh"

#define DEBUG_GPU_RR false

using namespace std;


class RQA_ConstParams {
public:
	static const int nRows_per_thread = 1;
	static const int warp = 32;
	static const int shared_memory_size = 513;
};

#define NTHREADS 256
#define LAM_NTHREADS 1024
#define Y_STEPS 4
#define X_STEPS 4
#define HALF_WARP 16
#define NSEEDS 32
#define WARP 32
#define BUFFER 32


template<class const_params, typename IOtype>
__global__ void GPU_RQA_RR_kernel(
		unsigned long long int d_RR_metric_integer, 
		IOtype const* __restrict__ d_input, 
		unsigned long long int size, 
		IOtype threshold, 
		int tau, 
		int emb
	){
	// Input data
	__shared__ int s_seeds[NSEEDS];
	__shared__ int s_sums[NTHREADS];
	extern __shared__ int s_local_RR[]; //local recurrent rate
	unsigned long long int pos_x, pos_y;
	
	//This checks if the threadblock is in the lower half of the R matrix
	//  (blockIdx.y*NSEEDS) represent beginning of the block within R matrix
	//  ((blockIdx.x+1)*NTHREADS - 1) represent end of the block within R matrix 
	//  if beginning of the block at y is greater then end of the block in x then the block
	//  does not have any points on the diagonal or in the upper half of the R matrix
	if( (blockIdx.y*NSEEDS) > ((blockIdx.x+1)*NTHREADS - 1) ) return;
	
	s_sums[threadIdx.x] = 0;
	if( threadIdx.x < NSEEDS ) s_seeds[threadIdx.x] = 0;
	pos_x = blockIdx.x*NTHREADS + threadIdx.x;
	pos_y = blockIdx.y*NSEEDS + threadIdx.x;
	
	// i-th row from the R matrix; each thread iterates through these values
	if( threadIdx.x<NSEEDS && pos_y<size ) {
		s_seeds[threadIdx.x] = pos_y;
	}
	
	int sum = 0;
	if(pos_x<size){
		for(int f=0; f<NSEEDS; f++){
			pos_y = blockIdx.y*NSEEDS + f;
			// We process only upper triangle of the R matrix which the block may cover partially; 
			// this contribution is added twice since lower triangle is the same
			if( pos_y<pos_x && pos_y<size ) {
				//int result = R_element_cartesian(s_seeds[f], elements, threshold); 
				int result = R_element_max(d_input, s_seeds[f], pos_x, threshold, tau, emb, size);
				//int result = R_element_equ(d_input, s_seeds[f], pos_x, threshold, tau, emb, size);
				sum = sum + 2*result;
			}
			else if( pos_y == pos_x ){ // diagonal
				//int result = R_element_cartesian(d_input[pos_y], d_input[pos_x], threshold);
				int result = R_element_max(d_input, pos_y, pos_x, threshold, tau, emb, size);
				//int result = R_element_equ(d_input, pos_y, pos_x, threshold, tau, emb, size);
				sum = sum + result;
				
			}
		}
		
		s_sums[threadIdx.x] = sum;
		__syncthreads();
		sum = Reduce_SM(s_sums);
		Reduce_WARP(&sum);
		__syncthreads();
		if(threadIdx.x==0) s_local_RR[t] = sum;
	}
	
	__syncthreads();
	

}
// ***********************************************************************************
// ***********************************************************************************
// ***********************************************************************************

template<class const_params, typename IOtype>
int RQA_RR_GPU_sharedmemory_metric(
		unsigned long long int d_RR_metric_integer, 
		IOtype *d_input, 
		unsigned long long int corrected_size, 
		IOtype threshold, 
		int tau, 
		int emb, 
		double *exec_time
	){
	GpuTimer timer;
	
	//---------> Task specific
	int nBlocks_x, nBlocks_y;

	nBlocks_x = (corrected_size + NTHREADS - 1)/(NTHREADS);
	nBlocks_y = (corrected_size + NSEEDS - 1)/(NSEEDS);
	
	dim3 gridSize(nBlocks_x, nBlocks_y, 1);
	dim3 blockSize(NTHREADS, 1, 1);
	
	if(DEBUG) printf("Data dimensions: %llu;\n",corrected_size);
	if(DEBUG) printf("Grid  settings: x:%d; y:%d; z:%d;\n", gridSize.x, gridSize.y, gridSize.z);
	if(DEBUG) printf("Block settings: x:%d; y:%d; z:%d;\n", blockSize.x, blockSize.y, blockSize.z);
	
	// ----------------------------------------------->
	// --------> Measured part
	timer.Start();
	
	//---------> Kernel execution
	RQA_R_init();
	GPU_RQA_RR_kernel<const_params><<< gridSize , blockSize, 1*sizeof(int)>>>(d_RR_metric_integer, d_input, corrected_size, threshold, tau, emb);
	
	timer.Stop();
	*exec_time += timer.Elapsed();
	// --------> Measured part
	// ----------------------------------------------->
	return(0);
}

int check_memory(size_t total_size, float multiple){
	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	double free_memory     = ((double) free_mem);
	double required_memory = multiple*((double) total_size);
	if(DEBUG) printf("\n");
	if(DEBUG) printf("Device has %0.3f MB of total memory, which %0.3f MB is available. Memory required %0.3f MB\n", ((float) total_mem)/(1024.0*1024.0), free_memory/(1024.0*1024.0), required_memory/(1024.0*1024.0));
	if(required_memory>free_memory) {printf("\n \n Array is too big for the device! \n \n"); return(3);}
	return(0);
}


template<class const_params, typename IOtype>
int GPU_RQA_RR_metric_tp(
		unsigned long long int *h_RR_metric_integer, 
		IOtype *h_input, 
		long int input_size, 
		IOtype threshold,
		int tau, 
		int emb, 
		int device, 
		double *execution_time
	){
	//---------> Initial nVidia stuff
	int devCount;
	checkCudaErrors(hipGetDeviceCount(&devCount));
	if(device<devCount) checkCudaErrors(hipSetDevice(device));
	else { printf("Wrong device!\n"); exit(1); }
	
	//---------> Checking memory
	size_t total_size = input_size*sizeof(IOtype);
	if(check_memory(total_size, 1.0)!=0) return(1);
	
	//---------> Measurements
	double exec_time = 0;
	GpuTimer timer;

	//---------> Memory allocation
	if (DEBUG) printf("Device memory allocation...: \t\t");
	long int corrected_size = input_size - (emb - 1)*tau;
	size_t input_size_bytes = input_size*sizeof(IOtype);
	IOtype *d_input;
	IOtype d_threshold;
	unsigned long long int d_RR_metric_integer;
	timer.Start();
	checkCudaErrors(hipMalloc((void **) &d_input, input_size_bytes) );
	timer.Stop();
	if (DEBUG) printf("done in %g ms.\n", timer.Elapsed());

	//---------> RR calculation
		//-----> Copy chunk of input data to a device
		checkCudaErrors(hipMemcpy(d_input, h_input, input_size_bytes, hipMemcpyHostToDevice));
		//-----> Compute RR
		
		RQA_RR_GPU_sharedmemory_metric<RQA_ConstParams>(d_RR_metric_integer, d_input, corrected_size, threshold, tau, emb, &exec_time);
		
		*execution_time = exec_time;
		if(DEBUG) printf("RQA recurrent rate: %f;\n", exec_time);
		
		checkCudaErrors(hipGetLastError());
		
		//-----> Copy chunk of output data to host
		checkCudaErrors(hipMemcpy(h_RR_metric_integer, d_RR_metric_integer, 1*sizeof(unsigned long long int), hipMemcpyDeviceToHost));
	//------------------------------------<
		
	//---------> error check -----
	checkCudaErrors(hipGetLastError());
	
	//---------> Feeing allocated resources
	checkCudaErrors(hipFree(d_input));
	
	return(0);
}

template<class const_params, typename IOtype>
void RQA_GPU_RR_metric_batch_runner(
		unsigned long long int *h_RR_metric_integer, 
		IOtype *h_input, 
		size_t input_size, 
		IOtype threshold,
		int tau, 
		int emb, 
		int device, 
		double *total_execution_time
	){
	// calculate RR
	double execution_time = 0;
	*total_execution_time = 0;
	GPU_RQA_RR_metric_tp<const_params,IOtype>(
			temp_rr_count, // 
			h_input, 
			input_size, 
			threshold,
			tau, 
			emb, 
			device, 
			&execution_time
		);

	(*total_execution_time) = (*total_execution_time) + execution_time;
	
	if(DEBUG_GPU_RR){
		long int corrected_size = input_size - (emb - 1)*tau;
		printf("DEBUG_GPU_RR: Temporary output:\n");
		printf("DEBUG_GPU_RR:   ");
		for(int i=0; i<(int)(th_chunks[f] + 1); i++){
			printf("%e  ", ((double) temp_rr_count[i])/((double) (corrected_size*corrected_size)) );
		}
		printf("\n");
	}
	
	// copy results to global results
	memcpy(&h_RR_metric_integer[th_shift], &temp_rr_count[1], th_chunks[f]*sizeof(unsigned long long int));


}

//-------------------------------------------------->
//------------ Wrappers for templating 


int GPU_RQA_RR_metric(unsigned long long int *h_RR_metric_integer, double *h_input, size_t input_size, double threshold, int tau, int emb, int distance_type, int device, double *execution_time){
	RQA_GPU_RR_metric_batch_runner<RQA_ConstParams, double>(h_RR_metric_integer, h_input, input_size, threshold, tau, emb, device, execution_time);
	return(0);
}

int GPU_RQA_RR_metric(unsigned long long int *h_RR_metric_integer, float *h_input, size_t input_size, float threshold, int tau, int emb, int distance_type, int device, double *execution_time){
	RQA_GPU_RR_metric_batch_runner<RQA_ConstParams, float>(h_RR_metric_integer, h_input, input_size, threshold, tau, emb, device, execution_time);
	return(0);
}

//---------------------------------------------------<